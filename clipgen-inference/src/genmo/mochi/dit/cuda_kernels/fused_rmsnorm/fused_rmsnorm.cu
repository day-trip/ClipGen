#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAStream.h>
#include <c10/cuda/CUDAException.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <>
#include <cmath>

#define WARP_SIZE 32
#define FULL_MASK 0xffffffffu

// Warp reduce (sum)
__device__ __forceinline__ float warp_reduce_sum(float val) {
    #pragma unroll
    for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(FULL_MASK, val, offset);
    }
    return val;
}

// Block reduce with broadcast to all threads
__device__ __forceinline__ float block_reduce_sum_broadcast(float val, float* shared) {
    const int lane = threadIdx.x & (WARP_SIZE - 1);
    const int wid  = threadIdx.x >> 5; // warp id in block
    const int nwarps = (blockDim.x + WARP_SIZE - 1) / WARP_SIZE;

    // Reduce within warp
    val = warp_reduce_sum(val);

    // Write each warp's sum
    if (lane == 0) {
        shared[wid] = val;
    }
    __syncthreads();

    // Final reduction in warp 0
    float block_sum = 0.f;
    if (wid == 0) {
        float warp_val = (threadIdx.x < nwarps) ? shared[lane] : 0.f;
        block_sum = warp_reduce_sum(warp_val);
        if (lane == 0) shared[0] = block_sum; // broadcast
    }
    __syncthreads();
    return shared[0];
}

// General kernel, handles any D
template <bool ExactMode>
__global__ void fused_residual_tanh_gated_rmsnorm_kernel(
    const __hip_bfloat16* __restrict__ x,       // [N, D]
    const __hip_bfloat16* __restrict__ x_res,   // [N, D]
    const __hip_bfloat16* __restrict__ gate,    // [N]
    __hip_bfloat16* __restrict__ out,           // [N, D]
    const int N,
    const int D,
    const float eps
) {
    const int row = blockIdx.x;
    if (row >= N) return;

    // shared layout: [nwarps floats for reduction] + [2 scalars: gate, inv_rms]
    const int nwarps = (blockDim.x + WARP_SIZE - 1) / WARP_SIZE;
    extern __shared__ float smem[];
    float* s_reduce = smem;
    float* s_scalars = smem + nwarps; // s_scalars[0] = gate_tanh, s_scalars[1] = inv_rms

    // Thread 0 computes tanh(gate) once
    if (threadIdx.x == 0) {
        float g = __bfloat162float(gate[row]);
        g = tanhf(g);
        if (ExactMode) {
            g = __bfloat162float(__float2bfloat16(g));
        }
        s_scalars[0] = g;
    }
    __syncthreads();

    const int base = row * D;

    // Phase 1: sum of squares in FP32
    float thread_sum = 0.f;
    for (int idx = threadIdx.x; idx < D; idx += blockDim.x) {
        float v = __bfloat162float(x_res[base + idx]);
        thread_sum = fmaf(v, v, thread_sum);
    }

    const float total_sum = block_reduce_sum_broadcast(thread_sum, s_reduce);

    // Thread 0 computes inv_rms once
    if (threadIdx.x == 0) {
        const float mean_sq = total_sum / static_cast<float>(D);
        s_scalars[1] = rsqrtf(mean_sq + eps);
    }
    __syncthreads();

    const float gate_tanh = s_scalars[0];
    const float inv_rms   = s_scalars[1];
    const float scale     = inv_rms * gate_tanh;

    // Phase 2: normalize, gate, add residual (with optional exact-mode rounding)
    if (ExactMode) {
        for (int idx = threadIdx.x; idx < D; idx += blockDim.x) {
            float xv  = __bfloat162float(x[base + idx]);
            float xrv = __bfloat162float(x_res[base + idx]);
            float x_normed_f = xrv * scale;
            __hip_bfloat16 x_normed_b = __float2bfloat16(x_normed_f);
            // Simulate bf16+bf16 add then round to bf16
            float sum_f = __bfloat162float(__float2bfloat16(xv)) + __bfloat162float(x_normed_b);
            out[base + idx] = __float2bfloat16(sum_f);
        }
    } else {
        for (int idx = threadIdx.x; idx < D; idx += blockDim.x) {
            float xv  = __bfloat162float(x[base + idx]);
            float xrv = __bfloat162float(x_res[base + idx]);
            float res = fmaf(xrv, scale, xv);
            out[base + idx] = __float2bfloat16(res);
        }
    }
}

// Specialized kernel for D = 3072 (6 elems per thread at 512 threads)
template <bool ExactMode>
__global__ void fused_residual_tanh_gated_rmsnorm_kernel_3072(
    const __hip_bfloat16* __restrict__ x,       // [N, 3072]
    const __hip_bfloat16* __restrict__ x_res,   // [N, 3072]
    const __hip_bfloat16* __restrict__ gate,    // [N]
    __hip_bfloat16* __restrict__ out,           // [N, 3072]
    const int N,
    const float eps
) {
    constexpr int D = 3072;
    constexpr int THREADS = 512;

    const int row = blockIdx.x;
    if (row >= N) return;

    // shared layout: [THREADS/32 floats for reduction] + [2 scalars: gate, inv_rms]
    __shared__ float s_reduce[THREADS / WARP_SIZE];
    __shared__ float s_scalars[2];

    // Thread 0 computes tanh(gate)
    if (threadIdx.x == 0) {
        float g = __bfloat162float(gate[row]);
        g = tanhf(g);
        if (ExactMode) {
            g = __bfloat162float(__float2bfloat16(g));
        }
        s_scalars[0] = g;
    }
    __syncthreads();

    const int base = row * D;
    const int tid = threadIdx.x;

    // Each thread handles 6 elements
    float xv[6], xrv[6];
    float thread_sum = 0.f;

    #pragma unroll
    for (int i = 0; i < 6; ++i) {
        const int idx = tid + i * THREADS;
        xrv[i] = __bfloat162float(x_res[base + idx]);
        xv[i]  = __bfloat162float(x[base + idx]);
        thread_sum = fmaf(xrv[i], xrv[i], thread_sum);
    }

    // Reduce sum to all threads
    // First reduce within warp
    float warp_sum = warp_reduce_sum(thread_sum);
    // Write warp results
    if ((tid & (WARP_SIZE - 1)) == 0) {
        s_reduce[tid / WARP_SIZE] = warp_sum;
    }
    __syncthreads();

    // Final reduction in warp 0 and broadcast
    if ((tid / WARP_SIZE) == 0) {
        float block_sum = (tid < THREADS / WARP_SIZE) ? s_reduce[tid] : 0.f;
        block_sum = warp_reduce_sum(block_sum);
        if ((tid & (WARP_SIZE - 1)) == 0) s_scalars[1] = rsqrtf(block_sum / float(D) + eps);
    }
    __syncthreads();

    const float scale = s_scalars[0] * s_scalars[1];

    if (ExactMode) {
        #pragma unroll
        for (int i = 0; i < 6; ++i) {
            float x_normed_f = xrv[i] * scale;
            __hip_bfloat16 x_normed_b = __float2bfloat16(x_normed_f);
            float sum_f = __bfloat162float(__float2bfloat16(xv[i])) + __bfloat162float(x_normed_b);
            out[base + (tid + i * THREADS)] = __float2bfloat16(sum_f);
        }
    } else {
        #pragma unroll
        for (int i = 0; i < 6; ++i) {
            float res = fmaf(xrv[i], scale, xv[i]);
            out[base + (tid + i * THREADS)] = __float2bfloat16(res);
        }
    }
}

// Utility: next power-of-two not exceeding limit and >= 32
static inline int pick_block_threads(int D) {
    int t = 32;
    while (t < D && t < 512) t <<= 1;
    if (t > 512) t = 512;
    return t;
}

template <bool ExactMode>
static void launch_kernel_dispatch(
    const __hip_bfloat16* x,
    const __hip_bfloat16* x_res,
    const __hip_bfloat16* gate,
    __hip_bfloat16* out,
    int N,
    int D,
    float eps,
    hipStream_t stream
) {
    if (N == 0 || D == 0) return;

    // Special fast path for D=3072 with 512 threads
    if (D == 3072) {
        dim3 grid(N);
        dim3 block(512);
        fused_residual_tanh_gated_rmsnorm_kernel_3072<ExactMode>
            <<<grid, block, 0, stream>>>(x, x_res, gate, out, N, eps);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
        return;
    }

    // General path
    const int threads = pick_block_threads(D);
    const int nwarps = (threads + WARP_SIZE - 1) / WARP_SIZE;
    // shared: nwarps for reduce + 2 scalars
    size_t shmem = (nwarps + 2) * sizeof(float);

    dim3 grid(N);
    dim3 block(threads);

    fused_residual_tanh_gated_rmsnorm_kernel<ExactMode>
        <<<grid, block, shmem, stream>>>(x, x_res, gate, out, N, D, eps);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
}

// Host entry
torch::Tensor fused_residual_tanh_gated_rmsnorm_cuda(
    torch::Tensor x,         // [N, D], bf16
    torch::Tensor x_res,     // [N, D], bf16
    torch::Tensor gate,      // [N] or [N,1], bf16
    double eps_d = 1e-6,
    bool exact_mode = false
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be CUDA");
    TORCH_CHECK(x_res.device().is_cuda(), "x_res must be CUDA");
    TORCH_CHECK(gate.device().is_cuda(), "gate must be CUDA");

    TORCH_CHECK(x.scalar_type() == at::kBFloat16, "x must be bfloat16");
    TORCH_CHECK(x_res.scalar_type() == at::kBFloat16, "x_res must be bfloat16");
    TORCH_CHECK(gate.scalar_type() == at::kBFloat16, "gate must be bfloat16");

    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(x_res.is_contiguous(), "x_res must be contiguous");
    TORCH_CHECK(gate.is_contiguous(), "gate must be contiguous");

    TORCH_CHECK(x.dim() == 2 && x_res.dim() == 2, "x and x_res must be 2D");
    TORCH_CHECK(x.sizes() == x_res.sizes(), "x and x_res must have the same shape");

    const int64_t N64 = x.size(0);
    const int64_t D64 = x.size(1);
    TORCH_CHECK(N64 <= INT_MAX && D64 <= INT_MAX, "N or D too large");
    const int N = static_cast<int>(N64);
    const int D = static_cast<int>(D64);

    // Accept gate shape [N] or [N,1]; flatten
    TORCH_CHECK(gate.numel() == N, "gate must have N elements (shape [N] or [N,1])");
    auto gate_flat = gate.reshape({N}).contiguous();

    auto out = torch::empty_like(x);

    const __hip_bfloat16* x_ptr     = reinterpret_cast<const __hip_bfloat16*>(x.data_ptr<at::BFloat16>());
    const __hip_bfloat16* x_res_ptr = reinterpret_cast<const __hip_bfloat16*>(x_res.data_ptr<at::BFloat16>());
    const __hip_bfloat16* gate_ptr  = reinterpret_cast<const __hip_bfloat16*>(gate_flat.data_ptr<at::BFloat16>());
    __hip_bfloat16* out_ptr         = reinterpret_cast<__hip_bfloat16*>(out.data_ptr<at::BFloat16>());

    auto stream = at::cuda::getCurrentCUDAStream();
    const float eps = static_cast<float>(eps_d);

    if (exact_mode) {
        launch_kernel_dispatch<true>(x_ptr, x_res_ptr, gate_ptr, out_ptr, N, D, eps, stream);
    } else {
        launch_kernel_dispatch<false>(x_ptr, x_res_ptr, gate_ptr, out_ptr, N, D, eps, stream);
    }

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("fused_residual_tanh_gated_rmsnorm_cuda",
          &fused_residual_tanh_gated_rmsnorm_cuda,
          "Fused Residual Tanh-Gated RMSNorm (CUDA)",
          py::arg("x"),
          py::arg("x_res"),
          py::arg("gate"),
          py::arg("eps") = 1e-6,
          py::arg("exact_mode") = false);
}