#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <torch/library.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAStream.h>
#include <c10/cuda/CUDAException.h>
#include <ATen/ATen.h>
#include <ATen/core/Tensor.h>
#include <ATen/ops/empty_like.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <>
#include <cmath>

#define WARP_SIZE 32
#define FULL_MASK 0xffffffffu

// Warp reduce (sum)
__device__ __forceinline__ float warp_reduce_sum(float val) {
    #pragma unroll
    for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(FULL_MASK, val, offset);
    }
    return val;
}

// Block reduce with broadcast to all threads
__device__ __forceinline__ float block_reduce_sum_broadcast(float val, float* shared) {
    const int lane = threadIdx.x & (WARP_SIZE - 1);
    const int wid  = threadIdx.x >> 5; // warp id in block
    const int nwarps = (blockDim.x + WARP_SIZE - 1) / WARP_SIZE;

    // Reduce within warp
    val = warp_reduce_sum(val);

    // Write each warp's sum
    if (lane == 0) {
        shared[wid] = val;
    }
    __syncthreads();

    // Final reduction in warp 0
    float block_sum = 0.f;
    if (wid == 0) {
        float warp_val = (threadIdx.x < nwarps) ? shared[lane] : 0.f;
        block_sum = warp_reduce_sum(warp_val);
        if (lane == 0) shared[0] = block_sum; // broadcast
    }
    __syncthreads();
    return shared[0];
}

// General kernel: per-feature gating
template <bool ExactMode>
__global__ void fused_residual_tanh_gated_rmsnorm_kernel(
    const __hip_bfloat16* __restrict__ x,       // [N, D]
    const __hip_bfloat16* __restrict__ x_res,   // [N, D]
    const __hip_bfloat16* __restrict__ gate,    // [N, D]
    __hip_bfloat16* __restrict__ out,           // [N, D]
    const int N,
    const int D,
    const float eps
) {
    const int row = blockIdx.x;
    if (row >= N) return;

    // Shared memory for reduction only
    const int nwarps = (blockDim.x + WARP_SIZE - 1) / WARP_SIZE;
    extern __shared__ float s_reduce[]; // size = nwarps

    const int base = row * D;

    // Phase 1: sum of squares in FP32 (over x_res row)
    float thread_sum = 0.f;
    for (int idx = threadIdx.x; idx < D; idx += blockDim.x) {
        float v = __bfloat162float(x_res[base + idx]);
        thread_sum = fmaf(v, v, thread_sum);
    }
    const float total_sum = block_reduce_sum_broadcast(thread_sum, s_reduce);

    // Compute inv_rms once, broadcast via shared[0]
    if (threadIdx.x == 0) {
        const float mean_sq = total_sum / static_cast<float>(D);
        s_reduce[0] = rsqrtf(mean_sq + eps);
    }
    __syncthreads();
    const float inv_rms = s_reduce[0];

    // Phase 2: elementwise tanh(gate), scale and residual add
    if (ExactMode) {
        for (int idx = threadIdx.x; idx < D; idx += blockDim.x) {
            const int off = base + idx;

            // Load and quantize inputs as needed
            float xv  = __bfloat162float(x[off]);
            float xrv = __bfloat162float(x_res[off]);

            // Per-feature tanh gate (optionally quantize gate_tanh to bf16)
            float g = __bfloat162float(gate[off]);
            g = tanhf(g);
            g = __bfloat162float(__float2bfloat16(g)); // quantize tanh(gate) to bf16

            // Normalize x_res in fp32, then quantize to bf16 before the add
            float x_normed_f = xrv * (inv_rms * g);
            __hip_bfloat16 x_normed_b = __float2bfloat16(x_normed_f);

            // Simulate bf16 + bf16 add
            float sum_f = __bfloat162float(__float2bfloat16(xv)) + __bfloat162float(x_normed_b);
            out[off] = __float2bfloat16(sum_f);
        }
    } else {
        for (int idx = threadIdx.x; idx < D; idx += blockDim.x) {
            const int off = base + idx;

            float xv  = __bfloat162float(x[off]);
            float xrv = __bfloat162float(x_res[off]);

            float g = __bfloat162float(gate[off]);
            g = tanhf(g);

            float res = fmaf(xrv, inv_rms * g, xv);
            out[off] = __float2bfloat16(res);
        }
    }
}

// Specialized kernel for D = 3072 (6 elems per thread at 512 threads), per-feature gating
template <bool ExactMode>
__global__ void fused_residual_tanh_gated_rmsnorm_kernel_3072(
    const __hip_bfloat16* __restrict__ x,       // [N, 3072]
    const __hip_bfloat16* __restrict__ x_res,   // [N, 3072]
    const __hip_bfloat16* __restrict__ gate,    // [N, 3072]
    __hip_bfloat16* __restrict__ out,           // [N, 3072]
    const int N,
    const float eps
) {
    constexpr int D = 3072;
    constexpr int THREADS = 512;

    const int row = blockIdx.x;
    if (row >= N) return;

    __shared__ float s_reduce[THREADS / WARP_SIZE]; // 16 floats
    __shared__ float s_inv_rms;

    const int base = row * D;
    const int tid = threadIdx.x;

    // Each thread handles 6 elements
    float xv[6], xrv[6], gv[6];
    float thread_sum = 0.f;

    #pragma unroll
    for (int i = 0; i < 6; ++i) {
        const int idx = tid + i * THREADS;
        const int off = base + idx;
        xrv[i] = __bfloat162float(x_res[off]);
        xv[i]  = __bfloat162float(x[off]);

        float g = __bfloat162float(gate[off]);
        g = tanhf(g);
        if (ExactMode) {
            g = __bfloat162float(__float2bfloat16(g)); // quantize tanh(gate) to bf16
        }
        gv[i] = g;

        thread_sum = fmaf(xrv[i], xrv[i], thread_sum);
    }

    // Reduce sum to all threads
    float warp_sum = warp_reduce_sum(thread_sum);
    if ((tid & (WARP_SIZE - 1)) == 0) {
        s_reduce[tid / WARP_SIZE] = warp_sum;
    }
    __syncthreads();

    // Final reduction in warp 0 and broadcast inv_rms
    if ((tid / WARP_SIZE) == 0) {
        float block_sum = (tid < THREADS / WARP_SIZE) ? s_reduce[tid] : 0.f;
        block_sum = warp_reduce_sum(block_sum);
        if ((tid & (WARP_SIZE - 1)) == 0) s_inv_rms = rsqrtf(block_sum / float(D) + eps);
    }
    __syncthreads();

    const float inv_rms = s_inv_rms;

    if (ExactMode) {
        #pragma unroll
        for (int i = 0; i < 6; ++i) {
            float x_normed_f = xrv[i] * (inv_rms * gv[i]);
            __hip_bfloat16 x_normed_b = __float2bfloat16(x_normed_f);

            float sum_f = __bfloat162float(__float2bfloat16(xv[i])) + __bfloat162float(x_normed_b);
            out[base + (tid + i * THREADS)] = __float2bfloat16(sum_f);
        }
    } else {
        #pragma unroll
        for (int i = 0; i < 6; ++i) {
            float res = fmaf(xrv[i], inv_rms * gv[i], xv[i]);
            out[base + (tid + i * THREADS)] = __float2bfloat16(res);
        }
    }
}

// Utility: next power-of-two not exceeding limit and >= 32
static inline int pick_block_threads(int D) {
    int t = 32;
    while (t < D && t < 512) t <<= 1;
    if (t > 512) t = 512;
    return t;
}

template <bool ExactMode>
static void launch_kernel_dispatch(
    const __hip_bfloat16* x,
    const __hip_bfloat16* x_res,
    const __hip_bfloat16* gate,
    __hip_bfloat16* out,
    int N,
    int D,
    float eps,
    hipStream_t stream
) {
    if (N == 0 || D == 0) return;

    // Specialized fast path for D=3072 with 512 threads
    if (D == 3072) {
        dim3 grid(N);
        dim3 block(512);
        fused_residual_tanh_gated_rmsnorm_kernel_3072<ExactMode>
            <<<grid, block, 0, stream>>>(x, x_res, gate, out, N, eps);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
        return;
    }

    // General path
    const int threads = pick_block_threads(D);
    const int nwarps = (threads + WARP_SIZE - 1) / WARP_SIZE;
    // shared: nwarps floats for reduction
    size_t shmem = nwarps * sizeof(float);

    dim3 grid(N);
    dim3 block(threads);

    fused_residual_tanh_gated_rmsnorm_kernel<ExactMode>
        <<<grid, block, shmem, stream>>>(x, x_res, gate, out, N, D, eps);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
}

// Host entry
torch::Tensor fused_residual_tanh_gated_rmsnorm_cuda(
    const torch::Tensor& x,       // [N, D], bf16
    const torch::Tensor& x_res,   // [N, D], bf16
    const torch::Tensor& gate,    // [N, D], bf16 (per-feature gate)
    double eps_d,
    bool exact_mode
) {
    TORCH_CHECK(x.device().is_cuda(), "x must be CUDA");
    TORCH_CHECK(x_res.device().is_cuda(), "x_res must be CUDA");
    TORCH_CHECK(gate.device().is_cuda(), "gate must be CUDA");

    TORCH_CHECK(x.scalar_type() == at::kBFloat16, "x must be bfloat16");
    TORCH_CHECK(x_res.scalar_type() == at::kBFloat16, "x_res must be bfloat16");
    TORCH_CHECK(gate.scalar_type() == at::kBFloat16, "gate must be bfloat16");

    TORCH_CHECK(x.is_contiguous(), "x must be contiguous");
    TORCH_CHECK(x_res.is_contiguous(), "x_res must be contiguous");
    TORCH_CHECK(gate.is_contiguous(), "gate must be contiguous");

    TORCH_CHECK(x.dim() == 2 && x_res.dim() == 2 && gate.dim() == 2, "x, x_res, gate must be 2D");
    TORCH_CHECK(x.sizes() == x_res.sizes(), "x and x_res must have the same shape");
    TORCH_CHECK(gate.sizes() == x.sizes(), "gate must have the same shape as x (per-feature)");

    const int64_t N64 = x.size(0);
    const int64_t D64 = x.size(1);
    TORCH_CHECK(N64 <= INT_MAX && D64 <= INT_MAX, "N or D too large");
    const int N = static_cast<int>(N64);
    const int D = static_cast<int>(D64);

    auto out = torch::empty_like(x);

    const __hip_bfloat16* x_ptr     = reinterpret_cast<const __hip_bfloat16*>(x.data_ptr<at::BFloat16>());
    const __hip_bfloat16* x_res_ptr = reinterpret_cast<const __hip_bfloat16*>(x_res.data_ptr<at::BFloat16>());
    const __hip_bfloat16* gate_ptr  = reinterpret_cast<const __hip_bfloat16*>(gate.data_ptr<at::BFloat16>());
    __hip_bfloat16* out_ptr         = reinterpret_cast<__hip_bfloat16*>(out.data_ptr<at::BFloat16>());

    auto stream = at::cuda::getCurrentCUDAStream();
    const float eps = static_cast<float>(eps_d);

    if (exact_mode) {
        launch_kernel_dispatch<true>(x_ptr, x_res_ptr, gate_ptr, out_ptr, N, D, eps, stream);
    } else {
        launch_kernel_dispatch<false>(x_ptr, x_res_ptr, gate_ptr, out_ptr, N, D, eps, stream);
    }

    return out;
}

// Meta kernel: validate shapes/dtypes and return a meta tensor
static at::Tensor fused_residual_tanh_gated_rmsnorm_meta(
const at::Tensor& x,
const at::Tensor& x_res,
const at::Tensor& gate,
double eps,
bool exact_mode) {

TORCH_CHECK(x.scalar_type() == at::kBFloat16, "x must be bfloat16");
TORCH_CHECK(x_res.scalar_type() == at::kBFloat16, "x_res must be bfloat16");
TORCH_CHECK(gate.scalar_type() == at::kBFloat16, "gate must be bfloat16");

TORCH_CHECK(x.dim() == 2 && x_res.dim() == 2 && gate.dim() == 2,
            "x, x_res, gate must be 2D [N, D] (per-feature gate)");
TORCH_CHECK(x.sizes() == x_res.sizes(), "x and x_res must have the same shape");
TORCH_CHECK(x.sizes() == gate.sizes(), "gate must have the same shape as x");

return at::empty_like(x, x.options().device(c10::Device(c10::kMeta)));
}

// Register op schema
TORCH_LIBRARY(mochi, m) {
m.def("fused_residual_tanh_gated_rmsnorm(Tensor x, Tensor x_res, Tensor gate, float eps=1e-6, bool exact_mode=False) -> Tensor");
}

// Register Meta and CUDA implementations
TORCH_LIBRARY_IMPL(mochi, Meta, m) {
m.impl("fused_residual_tanh_gated_rmsnorm", TORCH_FN(fused_residual_tanh_gated_rmsnorm_meta));
}

TORCH_LIBRARY_IMPL(mochi, CUDA, m) {
m.impl("fused_residual_tanh_gated_rmsnorm", TORCH_FN(fused_residual_tanh_gated_rmsnorm_cuda));
}

// Keep an empty pybind11 module so importing the extension loads this .so
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {}