#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <torch/library.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAStream.h>
#include <c10/cuda/CUDAException.h>
#include <ATen/ATen.h>
#include <ATen/core/Tensor.h>
#include <ATen/ops/empty_like.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <>
#include <cmath>
#include <tuple>

// Specialized kernel for D=1152 (4 elems per thread at 288 threads)
template <bool UpdateY>
__global__ void fused_conditioning_kernel_1152(
    const __hip_bfloat16* __restrict__ c,           // [B, 1152]
    const __hip_bfloat16* __restrict__ mod_x_weight, // [4608, 1152] (4*D, D)
    const __hip_bfloat16* __restrict__ mod_x_bias,   // [4608]
    const __hip_bfloat16* __restrict__ mod_y_weight, // [4608, 1152] or [1152, 1152]
    const __hip_bfloat16* __restrict__ mod_y_bias,   // [4608] or [1152]
    __hip_bfloat16* __restrict__ scale_msa_x,       // [B, 1152]
    __hip_bfloat16* __restrict__ gate_msa_x,        // [B, 1152]
    __hip_bfloat16* __restrict__ scale_mlp_x,       // [B, 1152]
    __hip_bfloat16* __restrict__ gate_mlp_x,        // [B, 1152]
    __hip_bfloat16* __restrict__ scale_msa_y,       // [B, 1152]
    __hip_bfloat16* __restrict__ gate_msa_y,        // [B, 1152] (if UpdateY)
    __hip_bfloat16* __restrict__ scale_mlp_y,       // [B, 1152] (if UpdateY)
    __hip_bfloat16* __restrict__ gate_mlp_y,        // [B, 1152] (if UpdateY)
    const int B
) {
    constexpr int D = 1152;
    constexpr int THREADS = 288;  // D / 4 = 288
    constexpr int OUT_DIM_X = 4 * D;  // 4608
    constexpr int OUT_DIM_Y = UpdateY ? 4 * D : D;

    const int batch_idx = blockIdx.x;
    if (batch_idx >= B) return;

    const int tid = threadIdx.x;

    // Shared memory for input c vector (reused for both linear layers)
    __shared__ float s_c[D];  // Store in fp32 for precision

    // Load c vector into shared memory with SiLU applied
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        const int idx = tid + i * THREADS;
        if (idx < D) {
            float c_val = __bfloat162float(c[batch_idx * D + idx]);
            // SiLU: x / (1 + exp(-x))
            s_c[idx] = c_val / (1.0f + expf(-c_val));
        }
    }
    __syncthreads();

    // Each thread computes 4 outputs for mod_x (16 total elements: 4 chunks × 4 elements)
    float mod_x_out[16];  // 4 outputs × 4 chunks

    #pragma unroll
    for (int chunk = 0; chunk < 4; ++chunk) {
        #pragma unroll
        for (int i = 0; i < 4; ++i) {
            const int out_idx = tid + i * THREADS + chunk * D;
            if (out_idx < OUT_DIM_X) {
                // GEMV: mod_x_weight[out_idx, :] · c + bias
                float acc = __bfloat162float(mod_x_bias[out_idx]);

                #pragma unroll
                for (int k = 0; k < D; ++k) {
                    float weight_val = __bfloat162float(mod_x_weight[out_idx * D + k]);
                    acc = fmaf(weight_val, s_c[k], acc);
                }

                mod_x_out[chunk * 4 + i] = acc;
            }
        }
    }

    // Store mod_x outputs to their respective chunks
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        const int idx = tid + i * THREADS;
        if (idx < D) {
            const int base_offset = batch_idx * D + idx;
            scale_msa_x[base_offset] = __float2bfloat16(mod_x_out[0 * 4 + i]);
            gate_msa_x[base_offset]  = __float2bfloat16(mod_x_out[1 * 4 + i]);
            scale_mlp_x[base_offset] = __float2bfloat16(mod_x_out[2 * 4 + i]);
            gate_mlp_x[base_offset]  = __float2bfloat16(mod_x_out[3 * 4 + i]);
        }
    }

    // Handle mod_y computation
    if (UpdateY) {
        // Similar pattern for mod_y with 4 chunks
        float mod_y_out[16];

        #pragma unroll
        for (int chunk = 0; chunk < 4; ++chunk) {
            #pragma unroll
            for (int i = 0; i < 4; ++i) {
                const int out_idx = tid + i * THREADS + chunk * D;
                if (out_idx < OUT_DIM_Y) {
                    float acc = __bfloat162float(mod_y_bias[out_idx]);

                    #pragma unroll
                    for (int k = 0; k < D; ++k) {
                        float weight_val = __bfloat162float(mod_y_weight[out_idx * D + k]);
                        acc = fmaf(weight_val, s_c[k], acc);
                    }

                    mod_y_out[chunk * 4 + i] = acc;
                }
            }
        }

        // Store mod_y outputs
        #pragma unroll
        for (int i = 0; i < 4; ++i) {
            const int idx = tid + i * THREADS;
            if (idx < D) {
                const int base_offset = batch_idx * D + idx;
                scale_msa_y[base_offset] = __float2bfloat16(mod_y_out[0 * 4 + i]);
                gate_msa_y[base_offset]  = __float2bfloat16(mod_y_out[1 * 4 + i]);
                scale_mlp_y[base_offset] = __float2bfloat16(mod_y_out[2 * 4 + i]);
                gate_mlp_y[base_offset]  = __float2bfloat16(mod_y_out[3 * 4 + i]);
            }
        }
    } else {
        // Just copy mod_y result to scale_msa_y (single output)
        #pragma unroll
        for (int i = 0; i < 4; ++i) {
            const int idx = tid + i * THREADS;
            if (idx < D) {
                float acc = __bfloat162float(mod_y_bias[idx]);

                #pragma unroll
                for (int k = 0; k < D; ++k) {
                    float weight_val = __bfloat162float(mod_y_weight[idx * D + k]);
                    acc = fmaf(weight_val, s_c[k], acc);
                }

                scale_msa_y[batch_idx * D + idx] = __float2bfloat16(acc);
            }
        }
    }
}

// Host entry
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor,
           torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
fused_conditioning_block_cuda(
    const torch::Tensor& c,              // [B, D], bf16
    const torch::Tensor& mod_x_weight,   // [4*D, D], bf16
    const torch::Tensor& mod_x_bias,     // [4*D], bf16
    const torch::Tensor& mod_y_weight,   // [4*D, D] or [D, D], bf16
    const torch::Tensor& mod_y_bias,     // [4*D] or [D], bf16
    bool update_y
) {
    TORCH_CHECK(c.device().is_cuda(), "c must be CUDA");
    TORCH_CHECK(c.scalar_type() == at::kBFloat16, "c must be bfloat16");
    TORCH_CHECK(c.is_contiguous(), "c must be contiguous");
    TORCH_CHECK(c.dim() == 2, "c must be 2D [B, D]");

    const int64_t B64 = c.size(0);
    const int64_t D64 = c.size(1);
    TORCH_CHECK(B64 <= INT_MAX && D64 <= INT_MAX, "B or D too large");
    const int B = static_cast<int>(B64);
    const int D = static_cast<int>(D64);

    // Create output tensors
    auto options = c.options();
    auto scale_msa_x = torch::empty({B, D}, options);
    auto gate_msa_x = torch::empty({B, D}, options);
    auto scale_mlp_x = torch::empty({B, D}, options);
    auto gate_mlp_x = torch::empty({B, D}, options);
    auto scale_msa_y = torch::empty({B, D}, options);

    torch::Tensor gate_msa_y, scale_mlp_y, gate_mlp_y;
    if (update_y) {
        gate_msa_y = torch::empty({B, D}, options);
        scale_mlp_y = torch::empty({B, D}, options);
        gate_mlp_y = torch::empty({B, D}, options);
    } else {
        gate_msa_y = torch::empty({0}, options);
        scale_mlp_y = torch::empty({0}, options);
        gate_mlp_y = torch::empty({0}, options);
    }

    // Get pointers
    const __hip_bfloat16* c_ptr = reinterpret_cast<const __hip_bfloat16*>(c.data_ptr<at::BFloat16>());
    const __hip_bfloat16* mod_x_weight_ptr = reinterpret_cast<const __hip_bfloat16*>(mod_x_weight.data_ptr<at::BFloat16>());
    const __hip_bfloat16* mod_x_bias_ptr = reinterpret_cast<const __hip_bfloat16*>(mod_x_bias.data_ptr<at::BFloat16>());
    const __hip_bfloat16* mod_y_weight_ptr = reinterpret_cast<const __hip_bfloat16*>(mod_y_weight.data_ptr<at::BFloat16>());
    const __hip_bfloat16* mod_y_bias_ptr = reinterpret_cast<const __hip_bfloat16*>(mod_y_bias.data_ptr<at::BFloat16>());

    __hip_bfloat16* scale_msa_x_ptr = reinterpret_cast<__hip_bfloat16*>(scale_msa_x.data_ptr<at::BFloat16>());
    __hip_bfloat16* gate_msa_x_ptr = reinterpret_cast<__hip_bfloat16*>(gate_msa_x.data_ptr<at::BFloat16>());
    __hip_bfloat16* scale_mlp_x_ptr = reinterpret_cast<__hip_bfloat16*>(scale_mlp_x.data_ptr<at::BFloat16>());
    __hip_bfloat16* gate_mlp_x_ptr = reinterpret_cast<__hip_bfloat16*>(gate_mlp_x.data_ptr<at::BFloat16>());
    __hip_bfloat16* scale_msa_y_ptr = reinterpret_cast<__hip_bfloat16*>(scale_msa_y.data_ptr<at::BFloat16>());
    __hip_bfloat16* gate_msa_y_ptr = update_y ? reinterpret_cast<__hip_bfloat16*>(gate_msa_y.data_ptr<at::BFloat16>()) : nullptr;
    __hip_bfloat16* scale_mlp_y_ptr = update_y ? reinterpret_cast<__hip_bfloat16*>(scale_mlp_y.data_ptr<at::BFloat16>()) : nullptr;
    __hip_bfloat16* gate_mlp_y_ptr = update_y ? reinterpret_cast<__hip_bfloat16*>(gate_mlp_y.data_ptr<at::BFloat16>()) : nullptr;

    auto stream = at::cuda::getCurrentCUDAStream();

    // Launch kernel (specialized for D=1152)
    if (D == 1152) {
        dim3 grid(B);
        dim3 block(288);  // D / 4
        if (update_y) {
            fused_conditioning_kernel_1152<true><<<grid, block, 0, stream>>>(
                c_ptr, mod_x_weight_ptr, mod_x_bias_ptr, mod_y_weight_ptr, mod_y_bias_ptr,
                scale_msa_x_ptr, gate_msa_x_ptr, scale_mlp_x_ptr, gate_mlp_x_ptr,
                scale_msa_y_ptr, gate_msa_y_ptr, scale_mlp_y_ptr, gate_mlp_y_ptr, B);
        } else {
            fused_conditioning_kernel_1152<false><<<grid, block, 0, stream>>>(
                c_ptr, mod_x_weight_ptr, mod_x_bias_ptr, mod_y_weight_ptr, mod_y_bias_ptr,
                scale_msa_x_ptr, gate_msa_x_ptr, scale_mlp_x_ptr, gate_mlp_x_ptr,
                scale_msa_y_ptr, gate_msa_y_ptr, scale_mlp_y_ptr, gate_mlp_y_ptr, B);
        }
    } else {
        TORCH_CHECK(false, "Only D=1152 supported currently");
    }

    C10_CUDA_KERNEL_LAUNCH_CHECK();

    return std::make_tuple(scale_msa_x, gate_msa_x, scale_mlp_x, gate_mlp_x,
                          scale_msa_y, gate_msa_y, scale_mlp_y, gate_mlp_y);
}

// Meta kernel: validate shapes/dtypes and return meta tensors
static std::tuple<at::Tensor, at::Tensor, at::Tensor, at::Tensor,
                  at::Tensor, at::Tensor, at::Tensor, at::Tensor>
fused_conditioning_block_meta(
    const at::Tensor& c,
    const at::Tensor& mod_x_weight,
    const at::Tensor& mod_x_bias,
    const at::Tensor& mod_y_weight,
    const at::Tensor& mod_y_bias,
    bool update_y) {

    TORCH_CHECK(c.scalar_type() == at::kBFloat16, "c must be bfloat16");
    TORCH_CHECK(c.dim() == 2, "c must be 2D [B, D]");

    const int64_t B = c.size(0);
    const int64_t D = c.size(1);
    auto options = c.options().device(c10::Device(c10::kMeta));

    auto scale_msa_x = at::empty({B, D}, options);
    auto gate_msa_x = at::empty({B, D}, options);
    auto scale_mlp_x = at::empty({B, D}, options);
    auto gate_mlp_x = at::empty({B, D}, options);
    auto scale_msa_y = at::empty({B, D}, options);
    auto gate_msa_y = update_y ? at::empty({B, D}, options) : at::empty({0}, options);
    auto scale_mlp_y = update_y ? at::empty({B, D}, options) : at::empty({0}, options);
    auto gate_mlp_y = update_y ? at::empty({B, D}, options) : at::empty({0}, options);

    return std::make_tuple(scale_msa_x, gate_msa_x, scale_mlp_x, gate_mlp_x,
                          scale_msa_y, gate_msa_y, scale_mlp_y, gate_mlp_y);
}

// Register op schema
TORCH_LIBRARY(mochi, m) {
    m.def("fused_conditioning_block(Tensor c, Tensor mod_x_weight, Tensor mod_x_bias, Tensor mod_y_weight, Tensor mod_y_bias, bool update_y) -> (Tensor, Tensor, Tensor, Tensor, Tensor, Tensor, Tensor, Tensor)");
}

// Register Meta and CUDA implementations
TORCH_LIBRARY_IMPL(mochi, Meta, m) {
    m.impl("fused_conditioning_block", TORCH_FN(fused_conditioning_block_meta));
}

TORCH_LIBRARY_IMPL(mochi, CUDA, m) {
    m.impl("fused_conditioning_block", TORCH_FN(fused_conditioning_block_cuda));
}

// Keep an empty pybind11 module so importing the extension loads this .so
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {}